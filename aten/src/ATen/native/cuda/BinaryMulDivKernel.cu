#include "hip/hip_runtime.h"
#include <ATen/AccumulateType.h>
#include <ATen/Dispatch.h>
#include <ATen/native/DispatchStub.h>
#include <ATen/native/cuda/Loops.cuh>
#include <ATen/native/TensorIterator.h>
#include <ATen/native/BinaryOps.h>
#include <c10/cuda/CUDAGuard.h>

// NOTE: CUDA on Windows requires that the enclosing function
// of a __device__ lambda not have internal linkage.

namespace at { namespace native {

void div_kernel_cuda(TensorIterator& iter) {
  if (!isIntegralType(iter.common_dtype(), /*includeBool*/ false) && iter.is_cpu_scalar(2)) {
    // optimization for floating-point types: if the second operand is a CPU
    // scalar, compute a * reciprocal(b). Note that this may lose one bit of
    // precision compared to computing the division.
    AT_DISPATCH_FLOATING_AND_COMPLEX_TYPES_AND2(kHalf, kBFloat16, iter.common_dtype(), "div_cuda", [&]() {
      using accscalar_t = at::acc_type<scalar_t, true>;
      auto inv_b = accscalar_t(1.0) / iter.scalar_value<accscalar_t>(2);
      iter.remove_operand(2);
      gpu_kernel(iter, [inv_b]GPU_LAMBDA(scalar_t a) -> scalar_t {
        return a * inv_b;
      });
    });
  } else {
    AT_DISPATCH_ALL_TYPES_AND_COMPLEX_AND2(kHalf, kBFloat16, iter.common_dtype(), "div_cuda", [&]() {
      gpu_kernel_with_scalars(iter, []GPU_LAMBDA(scalar_t a, scalar_t b) -> scalar_t {
        return a / b;
      });
    });
  }
}

void floordiv_integral_kernel_cuda(TensorIterator& iter) {
  // In the special case of unsigned integer division, floor division
  //   is equivalent to truncation division (since the signs of
  //   the divisor and dividend are always the same)
  if (iter.common_dtype() == c10::ScalarType::Byte) {
    gpu_kernel_with_scalars(iter, []GPU_LAMBDA(uint8_t a, uint8_t b) -> uint8_t {
      return a / b;
    });
  } else {
    AT_DISPATCH_INTEGRAL_TYPES(iter.common_dtype(), "floordiv_cuda", [&]() {
      gpu_kernel_with_scalars(iter, []GPU_LAMBDA(scalar_t a, scalar_t b) -> scalar_t {
        if ((a < 0) != (b < 0)) {
          // Subtracts one from the results of truncation division
          //   if the divisor and dividend have different sign(bit)s
          //   and the remainder of the division is nonzero
          const auto quot = a / b;
          const auto rem = a % b;
          return rem ? quot - 1 : quot;
        } else {
          // When the sign(bit)s of the divisor and dividend are the same
          //   truncation division is equivalent to floor division
          return a / b;
        }
      });
    });
  }
}

void mul_kernel_cuda(TensorIterator& iter) {
  if (iter.common_dtype() == ScalarType::Bool) {
    // Workaround for the error: '*' in boolean context, suggest '&&' instead [-Werror=int-in-bool-context]
    gpu_kernel_with_scalars(iter, []GPU_LAMBDA(bool a, bool b) -> bool {
      return a && b;
    });
  } else if (!isIntegralType(iter.common_dtype(), /*includeBool*/ false) &&
    (iter.is_cpu_scalar(1) || iter.is_cpu_scalar(2))) {
  //if common dtype is half the scalar constant can overflow in half precision, and yet the result can
  //still be representable in the half dtype. Cast scalar to acc_type to have better accuracy
          AT_DISPATCH_FLOATING_AND_COMPLEX_TYPES_AND2(kHalf, kBFloat16, iter.common_dtype(), "mul_cuda", [&]() {
            using accscalar_t = at::acc_type<scalar_t, true>;
            int scalar_arg = iter.is_cpu_scalar(1) ? 1 : 2;
            auto b = iter.scalar_value<accscalar_t>(scalar_arg);
            iter.remove_operand(scalar_arg);
            const cuda::OptionalCUDAGuard device_guard(device_of(iter.tensor(1)));
            gpu_kernel(iter, [b]GPU_LAMBDA(scalar_t a) -> scalar_t {
              return a * b;
            });
          });
  } else {
    AT_DISPATCH_ALL_TYPES_AND_COMPLEX_AND2(kHalf, kBFloat16, iter.common_dtype(), "mul_cuda", [&]() {
      gpu_kernel_with_scalars(iter, []GPU_LAMBDA(scalar_t a, scalar_t b) -> scalar_t {
        return a * b;
      });
    });
  }
}

REGISTER_DISPATCH(div_stub, &div_kernel_cuda);
REGISTER_DISPATCH(floordiv_integral_stub, &floordiv_integral_kernel_cuda);
REGISTER_DISPATCH(mul_stub, &mul_kernel_cuda);

}} // namespace at::native
